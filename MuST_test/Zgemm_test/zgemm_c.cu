// cuda_zgemm.cu
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include <iostream>

extern "C" void cuda_zgemm_c_(int* m, int* n, int* k,
                             const hipDoubleComplex* alpha,
                             const hipDoubleComplex* A, int* lda,
                             const hipDoubleComplex* B, int* ldb,
                             const hipDoubleComplex* beta,
                             hipDoubleComplex* C, int* ldc)
{

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Determine cuBLAS operation modes
    hipblasOperation_t cuTransA = HIPBLAS_OP_N;
    hipblasOperation_t cuTransB = HIPBLAS_OP_N;

    // Allocate device memory
    hipDoubleComplex *d_A, *d_B, *d_C;
    size_t size_A = (*lda) * (*k);
    size_t size_B = (*ldb) * (*n);
    size_t size_C = (*ldc) * (*n);
    printf("M = %d, N = %d, K = %d\n", *m, *n, *k);
    printf("LDA = %d, LDB = %d, LDC = %d\n", *lda, *ldb, *ldc);

    hipMalloc((void**)&d_A, sizeof(hipDoubleComplex) * size_A);
    hipMalloc((void**)&d_B, sizeof(hipDoubleComplex) * size_B);
    hipMalloc((void**)&d_C, sizeof(hipDoubleComplex) * size_C);

    // Copy data from host to device
    hipMemcpy(d_A, A, sizeof(hipDoubleComplex) * size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(hipDoubleComplex) * size_B, hipMemcpyHostToDevice);
    //cudaMemcpy(d_C, C, sizeof(cuDoubleComplex) * size_C, cudaMemcpyHostToDevice);

    // Perform ZGEMM operation
    hipblasStatus_t status = hipblasZgemm(handle, cuTransA, cuTransB,
                                      *m, *n, *k,
                                      alpha,
                                      d_A, *lda,
                                      d_B, *ldb,
                                      beta,
                                      d_C, *ldc);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "hipblasZgemm failed with error: " << status << std::endl;
    }

    // Copy result back to host
    hipMemcpy(C, d_C, sizeof(hipDoubleComplex) * size_C, hipMemcpyDeviceToHost);

    // Free device memory and destroy handle
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}
