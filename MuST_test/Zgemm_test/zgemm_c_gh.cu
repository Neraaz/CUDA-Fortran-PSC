#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include <iostream>

extern "C" void cuda_zgemm_c_(int* m, int* n, int* k,
                             const hipDoubleComplex* alpha,
                             const hipDoubleComplex* A, int* lda,
                             const hipDoubleComplex* B, int* ldb,
                             const hipDoubleComplex* beta,
                             hipDoubleComplex* C, int* ldc)
{
    // Set device to prefer unified memory
    hipSetDevice(0);
    cudaDeviceSetMemFlags(hipMemAttachGlobal);

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Determine cuBLAS operation modes
    hipblasOperation_t cuTransA = HIPBLAS_OP_N;
    hipblasOperation_t cuTransB = HIPBLAS_OP_N;

    // Allocate unified memory that's accessible from both CPU and GPU
    size_t size_A = (*lda) * (*k);
    size_t size_B = (*ldb) * (*n);
    size_t size_C = (*ldc) * (*n);
    
    printf("M = %d, N = %d, K = %d\n", *m, *n, *k);
    printf("LDA = %d, LDB = %d, LDC = %d\n", *lda, *ldb, *ldc);

    // Allocate unified memory
    hipDoubleComplex *u_A, *u_B, *u_C;
    hipMallocManaged(&u_A, sizeof(hipDoubleComplex) * size_A);
    hipMallocManaged(&u_B, sizeof(hipDoubleComplex) * size_B);
    hipMallocManaged(&u_C, sizeof(hipDoubleComplex) * size_C);

    // Copy data to unified memory (no need for explicit device copies)
    memcpy(u_A, A, sizeof(hipDoubleComplex) * size_A);
    memcpy(u_B, B, sizeof(hipDoubleComplex) * size_B);
    
    // Prefetch data to GPU for better performance
    hipMemPrefetchAsync(u_A, sizeof(hipDoubleComplex) * size_A, 0);
    hipMemPrefetchAsync(u_B, sizeof(hipDoubleComplex) * size_B, 0);
    hipMemPrefetchAsync(u_C, sizeof(hipDoubleComplex) * size_C, 0);

    // Perform ZGEMM operation
    hipblasStatus_t status = hipblasZgemm(handle, cuTransA, cuTransB,
                                      *m, *n, *k,
                                      alpha,
                                      u_A, *lda,
                                      u_B, *ldb,
                                      beta,
                                      u_C, *ldc);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "hipblasZgemm failed with error: " << status << std::endl;
    }

    // Wait for computation to finish
    hipDeviceSynchronize();

    // Copy result back (not strictly needed with unified memory, but maintains interface)
    memcpy(C, u_C, sizeof(hipDoubleComplex) * size_C);

    // Free unified memory and destroy handle
    hipFree(u_A);
    hipFree(u_B);
    hipFree(u_C);
    hipblasDestroy(handle);
}
